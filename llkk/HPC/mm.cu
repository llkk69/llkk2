#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void matadd(int *l, int *m, int *n) {
    int x = blockIdx.x;
    int y = blockIdx.y;
    int id = gridDim.x * y + x;
    n[id] = l[id] + m[id];
}

int main() {
    int a[2][3];
    int b[2][3];
    int c[2][3];
    int *d, *e, *f;
    int i, j;

    // Input for matrix A
    printf("Enter elements of first matrix of size 2 * 3:\n");
    for (i = 0; i < 2; i++) {
        for (j = 0; j < 3; j++) {
            scanf("%d", &a[i][j]);
        }
    }

    // Input for matrix B
    printf("Enter elements of second matrix of size 2 * 3:\n");
    for (i = 0; i < 2; i++) {
        for (j = 0; j < 3; j++) {
            scanf("%d", &b[i][j]);
        }
    }

    // Allocate device memory
    hipMalloc((void **)&d, 2 * 3 * sizeof(int));
    hipMalloc((void **)&e, 2 * 3 * sizeof(int));
    hipMalloc((void **)&f, 2 * 3 * sizeof(int));

    // Copy matrices A and B from host to device
    hipMemcpy(d, a, 2 * 3 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(e, b, 2 * 3 * sizeof(int), hipMemcpyHostToDevice);

    dim3 grid(3, 2);

    // Measure elapsed time
    hipEvent_t start, stop;
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    // Launch kernel for matrix addition
    matadd<<<grid, 1>>>(d, e, f);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calculate elapsed time
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Elapsed Time: %.6f ms\n", elapsedTime);

    // Copy the result matrix from device to host
    hipMemcpy(c, f, 2 * 3 * sizeof(int), hipMemcpyDeviceToHost);

    // Print the result matrix
    printf("\nSum of two matrices:\n");
    for (i = 0; i < 2; i++) {
        for (j = 0; j < 3; j++) {
            printf("%d\t", c[i][j]);
        }
        printf("\n");
    }

    // Free device memory
    hipFree(d);
    hipFree(e);
    hipFree(f);

    return 0;
}
