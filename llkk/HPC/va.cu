
#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void add(int* A, int* B, int* C, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size) {
        C[tid] = A[tid] + B[tid];
    }
}


void initialize(int* vector, int size) {
    for (int i = 0; i < size; i++) {
        vector[i] = rand() % 10;
    }
}

void print(int* vector, int size) {
    for (int i = 0; i < size; i++) {
        cout << vector[i] << " ";
    }
    cout << endl;
}

int main() {
    int N = 10;
    int* A, * B, * C;

    int vectorSize = N;
    size_t vectorBytes = vectorSize * sizeof(int);

    A = new int[vectorSize];
    B = new int[vectorSize];
    C = new int[vectorSize];

    initialize(A, vectorSize);
    initialize(B, vectorSize);

    cout << "Vector A: ";
    print(A, N);
    cout << "Vector B: ";
    print(B, N);

    // Allocate memory on host only
    int* d_A, * d_B, * d_C;
    hipMalloc(&d_A, vectorBytes);
    hipMalloc(&d_B, vectorBytes);
    hipMalloc(&d_C, vectorBytes);

    // Copy data from host to device
    hipMemcpy(d_A, A, vectorBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, vectorBytes, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Timing events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    // Kernel invocation
    add<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "Addition elapsed time: " << milliseconds << " ms" << endl;

    // Copy results back from device to host
    hipMemcpy(C, d_C, vectorBytes, hipMemcpyDeviceToHost);

    cout << "Addition: ";
    print(C, N);

    delete[] A;
    delete[] B;
    delete[] C;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
